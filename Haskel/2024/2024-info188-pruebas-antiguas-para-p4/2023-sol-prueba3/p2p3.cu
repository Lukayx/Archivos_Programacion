
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <omp.h>

// **************** COMPLETAR *****************
#define BX 8
#define BY 8
#define BZ 8
// ********************************************

// Filtro Gaussiano CUDA
__global__ void kernelFG(float *input, float *output, int width, int height, int depth) {
    // **************** COMPLETAR *****************
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int idz = threadIdx.z + blockIdx.z * blockDim.z;

    if (idx < width && idy < height && idz < depth) {
        long index = (long)idz * width * height + idy * width + idx;
        float sum = 0.0f;
        int count = 0;

        for (int z = -1; z <= 1; ++z) {
            for (int y = -1; y <= 1; ++y) {
                for (int x = -1; x <= 1; ++x) {
                    int nx = idx + x;
                    int ny = idy + y;
                    int nz = idz + z;

                    if (nx >= 0 && ny >= 0 && nz >= 0 && nx < width && ny < height && nz < depth) {
                        long neighborIndex = (long)nz * width * height + ny * width + nx;
                        sum += input[neighborIndex];
                        count++;
                    }
                }
            }
        }
        output[index] = sum / count;
    }
    // ********************************************
}

// Filtro Gaussiano OpenMP
void openmpFG(float *input, float *output, int width, int height, int depth) {
    // **************** COMPLETAR *****************
    #pragma omp parallel for
    for (int z = 0; z < depth; ++z) {
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                int index = z * width * height + y * width + x;
                float sum = 0.0f;
                int count = 0;

                for (int dz = -1; dz <= 1; ++dz) {
                    for (int dy = -1; dy <= 1; ++dy) {
                        for (int dx = -1; dx <= 1; ++dx) {
                            int nx = x + dx;
                            int ny = y + dy;
                            int nz = z + dz;

                            if (nx >= 0 && ny >= 0 && nz >= 0 && nx < width && ny < height && nz < depth) {
                                int neighborIndex = nz * width * height + ny * width + nx;
                                sum += input[neighborIndex];
                                count++;
                            }
                        }
                    }
                }
                output[index] = sum / count;
            }
        }
    }
    // ********************************************
}

int main(int argc, char *argv[]) {
    if (argc != 5) {
        std::cerr << "Ejecutar como: "<< argv[0] << " <width> <height> <depth> <nt>" << std::endl;
        return 1;
    }

    // Recibir argumentos
    const int width = std::atoi(argv[1]);
    const int height = std::atoi(argv[2]);
    const int depth = std::atoi(argv[3]);
    const int nt    = std::atoi(argv[4]);
    const long size = (long)width * (long)height * (long)depth;

    printf("\n");
    printf("Matrix de width=%i x height=%i x depth=%i\n", width, height, depth);
    printf("Bloques CUDA de %i x %i x %i\n", BX, BY, BZ);
    printf("OpenMP Threads = %i\n\n", nt);
    printf("VRAM GPU: %f GB\n", (double)size * sizeof(float) * 2.0 / 1e9);
    printf("RAM  CPU: %f GB\n\n", (double)size * sizeof(float) * 3.0 / 1e9);

    float *input = new float[size];
    float *output_CUDA = new float[size];
    float *output_OpenMP = new float[size];

    // Inicializar datos
    printf("Inicializando matrix 3D..........."); fflush(stdout);
    for (long i = 0; i < size; ++i) {
        input[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    printf("listo\n"); fflush(stdout);



    // **************** COMPLETAR *****************
    // configuracion de grid CUDA
    dim3 blockDim(BX, BY, BZ);
    dim3 gridDim((width + BX - 1)/BX, (height + BY - 1) / BY, (depth + BZ - 1)/BZ);
    // ********************************************

    // alocar memoria GPU
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, size * sizeof(float));
    hipMalloc((void**)&d_output, size * sizeof(float));

    // Copiar datos de Host a Device
    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    // Herramientas CUDA para medir tiempo
    hipEvent_t start_CUDA, stop_CUDA;
    hipEventCreate(&start_CUDA);
    hipEventCreate(&stop_CUDA);

    // Ejecutar Kernel GPU
    printf("Ejecutando Kernel................."); fflush(stdout);
    hipEventRecord(start_CUDA);
    kernelFG<<<gridDim, blockDim>>>(d_input, d_output, width, height, depth);
    printf("listo: "); fflush(stdout);

    // Terminar de medir tiempo
    hipEventRecord(stop_CUDA);
    hipEventSynchronize(stop_CUDA);
    float cuda_time = 0;
    hipEventElapsedTime(&cuda_time, start_CUDA, stop_CUDA);
    std::cout << cuda_time/1000.0f << " secs" << std::endl;

    // Copiar resultado de vuelta a Host
    hipMemcpy(output_CUDA, d_output, size * sizeof(float), hipMemcpyDeviceToHost);




    // OPENMP
    omp_set_num_threads(nt);
    printf("Ejecutando OpenMP................."); fflush(stdout);
    double start_OpenMP = omp_get_wtime();
    // Ejecutar funcion OpenMP
    openmpFG(input, output_OpenMP, width, height, depth);
    printf("listo: "); fflush(stdout);
    double end_OpenMP = omp_get_wtime();
    double openmp_time = end_OpenMP - start_OpenMP;
    std::cout << openmp_time << " secs" << std::endl;





    // Comparar resultados entre CPU y GPU 
    printf("Comparando Resultados CPU/GPU....."); fflush(stdout);
    bool coinciden = true;
    for (long i = 0; i < size; ++i) {
        if (std::fabs(output_CUDA[i] - output_OpenMP[i]) > 1e-5) {
            std::cerr << "Error!" << std::endl;
            coinciden = false;
            break;
        }
    }
    if(coinciden){
        printf("coinciden\n"); fflush(stdout);
    }

    // Liberar memoria
    delete[] input;
    delete[] output_CUDA;
    delete[] output_OpenMP;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}

