#include "hip/hip_runtime.h"
/*
 * ******** Pregunta Q2 GPU CUDA (3.0 pts) ********
 * El siguiente programa es un 'molde' para un automata celular simulado en GPU. Un automata celular es un sistema dinamico 
 * donde cada celda obedece a una regla fija que involucra una operacion con el valor de las celdas vecinas. En este caso, 
 * el molde esta hecho para un automata celular en 2-dimensiones, donde cada celda tiene un estado de 0 o 1. Para conocer 
 * el estado siguiente (en el tiempo t+1) de una celda, se debe leer los estados de ella misma y los de la vencidad, y 
 * realizar una operacion. Esto debe hacerse en cada celda simultaneamente, por esa razon es que se trabaja con dos 
 * punteros para evitar problemas de concurrencia.
 * Puede probar ejecutando el programa con ./q2 32 8 32 17 1000 0.137
 *    
 * a) (1.5 pts) Implemente el automata celular del juego de la vida en GPU. Este automata celular tiene las siguientes reglas:
 *      - Cada celda puede tener un estado muerto (0) o vivo (1).
 *      - Para cada celda:
            - Si esta viva y tiene dos o tres vecinos vivos --> VIVE AL PASO SIGUIENTE
            - Si esta muerta y tiene 3 vecinos vivos --> VIVE AL PASO SIGUIENTE
            - Cualquier otro caso --> MUERE AL PASO SIGUIENTE

            A B C
            D E F  procesar E --> vecinos son A B C D F G H I
            G H I
        - Las condiciones de borde son periodicas, es decir el vecino del extremo derecho es el del extremo izquierdo y 
          vice-versa. Lo mismo de forma vertical.

 * b) (1.5 pts) Mida el tiempo promedio de un paso de simulacion con n=8192 y prob=0.13 usando los tamanos de bloque: {1, 2, 4, 8, 16, 32}. 
        - Haga un grafico con tiempo (Y) vs tamano-bloque (X). ¿Que puede concluir del efecto del tamano del bloque en el rendimiento de la GPU?
 */

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <omp.h>
#include "tools.h"


// TRABAJAR AQUI
__global__ void kernel_CA(int n, int *in, int *out){
    // **********************************
    // a) ESCRIBA SU SOLUCION AQUI
    // **********************************


    // HELP1: para acceder a la posicion (x,y) debe usar la forma [y*n + x]
    // HELP2: almacene las posiciones de derecha x+1, izq x-1, abajo y+1 y arriba y-1, antes de combinarlas. Asi las esquinas se construyen mas facil.
    // HELP3: recuerde que son condiciones periodicas. La operaciones modulo puede servirle para tratar bien x+1, y+1, pero para x-1, y-1 
    //        hagalo con  condicional.
}



// IGNORAR 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


int main(int argc, char **argv){
    if(argc != 7){
        fprintf(stderr, "ejecutar como ./q2 n nt B seed steps prob\n\n");
        exit(EXIT_FAILURE);
    }
    // args
    int n       = atoi(argv[1]);
    int nt      = atoi(argv[2]);
    int B       = atoi(argv[3]);
    int seed    = atoi(argv[4]);
    int steps   = atoi(argv[5]);
    float prob  = atof(argv[6]);
    omp_set_num_threads(nt);
    printf("n=%i  B=%i  steps=%i\n", n, B, steps);
    
    // creando datos
    int *d1, *d2;
    int *A = new int[n*n];
    float timems;
    init_prob(n, A, 13, prob);
    gpuErrchk(hipMalloc(&d1, sizeof(int)*n*n));
    gpuErrchk(hipMalloc(&d2, sizeof(int)*n*n));
    gpuErrchk(hipMemcpy(d1, A, sizeof(int)*n*n, hipMemcpyHostToDevice));

    // ejecucion
    print_mat(n, A, "INPUT");
    printf("Press Enter...\n"); fflush(stdout); getchar();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 block(B,B,1);
    dim3 grid((n+B-1)/B, (n+B-1)/B, 1);
    for(int i=0; i<steps; ++i){
        printf("Simulacion step=%i........", i);
        hipEventRecord(start);

        // llamada al kernel
        kernel_CA<<<grid, block>>>(n, d1, d2);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        // tiempo 
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&timems, start, stop);
        printf("done: %f\n", timems/1000.0);

        // copiar y mostrar
        gpuErrchk(hipMemcpy(A, d2, sizeof(int)*n*n, hipMemcpyDeviceToHost));
        print_mat(n, A, "Estado del automata celular");
        printf("Press Enter...\n"); fflush(stdout); getchar();
        std::swap(d1, d2);
    }
}
